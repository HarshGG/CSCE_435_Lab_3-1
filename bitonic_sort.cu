#include "hip/hip_runtime.h"
/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* bitonic_sort_step_region = "bitonic_sort_step";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";

// float cudaMemcpy_host_to_device_time = 0.0f;
// float cudaMemcpy_device_to_host_time = 0.0f;
// float bitonic_sort_step_time = 0.0f;

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void bitonic_sort(float *values)
{
  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipEvent_t startEvent;
  hipEventCreate(&startEvent);
  hipEvent_t stopEvent;
  hipEventCreate(&stopEvent);

  hipMalloc((void**) &dev_values, size);
  
  //MEM COPY FROM HOST TO DEVICE
  CALI_MARK_BEGIN(cudaMemcpy_host_to_device);
  hipEventRecord(startEvent, 0);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&cudaMemcpy_host_to_device_time, startEvent, stopEvent);
  CALI_MARK_END(cudaMemcpy_host_to_device);


  hipEventRecord(startEvent, 0);
  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */
  
  int j, k;
  /* Major step */
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  
  hipDeviceSynchronize();
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&bitonic_sort_step_time, startEvent, stopEvent);

  effective_bandwidth_gb_s = ((2.0f * NUM_VALS * sizeof(float)) / (1 << 30)) / (bitonic_sort_step_time / 1000.0);
  printf("Effective Bandwidth: %.3f GB/s\n", effective_bandwidth_gb_s);

  //MEM COPY FROM DEVICE TO HOST
  CALI_MARK_BEGIN(cudaMemcpy_device_to_host);
  hipEventRecord(startEvent, 0);
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&cudaMemcpy_device_to_host_time, startEvent, stopEvent);
  CALI_MARK_END(cudaMemcpy_device_to_host);

  hipFree(dev_values);

  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
}

int main(int argc, char *argv[])
{
  CALI_CXX_MARK_FUNCTION;

  THREADS = atoi(argv[1]);
  NUM_VALS = atoi(argv[2]);
  BLOCKS = NUM_VALS / THREADS;

  printf("Number of threads: %d\n", THREADS);
  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of blocks: %d\n", BLOCKS);

  // Create caliper ConfigManager object
  cali::ConfigManager mgr;
  mgr.start();

  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);

  clock_t start = clock();
  bitonic_sort(values); /* Inplace */
  clock_t stop = clock();

  print_elapsed(start, stop);

  // Store results in these variables.
  float effective_bandwidth_gb_s;
  float bitonic_sort_step_time;
  float cudaMemcpy_host_to_device_time;
  float cudaMemcpy_device_to_host_time;
  int kernel_iterations;

  adiak::init(NULL);
  adiak::user();
  adiak::launchdate();
  adiak::libraries();
  adiak::cmdline();
  adiak::clustername();
  adiak::value("num_threads", THREADS);
  adiak::value("num_blocks", BLOCKS);
  adiak::value("num_vals", NUM_VALS);
  adiak::value("program_name", "cuda_bitonic_sort");
  adiak::value("datatype_size", sizeof(float));
  adiak::value("effective_bandwidth (GB/s)", effective_bandwidth_gb_s);
  adiak::value("bitonic_sort_step_time", bitonic_sort_step_time);
  adiak::value("cudaMemcpy_host_to_device_time", cudaMemcpy_host_to_device_time);
  adiak::value("cudaMemcpy_device_to_host_time", cudaMemcpy_device_to_host_time);
  adiak::value("kernel_iterations", kernel_iterations);

  // Flush Caliper output
  mgr.stop();
  mgr.flush();

  array_print(values, NUM_VALS);
}
